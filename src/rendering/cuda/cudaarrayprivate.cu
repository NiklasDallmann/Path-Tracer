#include "rendering/cuda/cudaarrayprivate.h"

bool allocateManagedCudaMemory(void **data, const uint32_t size)
{
	bool returnValue = true;
	
	if (hipMallocManaged(data, size) != hipSuccess)
	{
		returnValue = false;
	}
	
	return returnValue;
}

bool freeCudaMemory(void *data)
{
	bool returnValue = true;
	
	if (hipFree(data) != hipSuccess)
	{
		returnValue = false;
	}
	
	return returnValue;
}
