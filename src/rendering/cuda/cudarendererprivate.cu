#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

//#include "abstractrenderer.h"
#include "cuda/cudaarray.h"
#include "cuda/cudatypes.h"
#include "rendering/framebuffer.h"
#include "math/algorithms.h"
#include "math/matrix4x4.h"
#include "math/vector4.h"
#include "rendering/ray.h"
#include "rendering/randomnumbergenerator.h"
#include "rendering/shader.h"

namespace ToyPT
{
namespace Rendering
{
namespace Cuda
{

__host__ void handleCudaError(const hipError_t error)
{
	if (error != hipSuccess)
	{
		printf("CUDA Error: %s %s\n", hipGetErrorName(error), hipGetErrorString(error));
	}
}

__device__ void print(const char *name, const Math::Vector4 &vector)
{
	printf("%s := {%f, %f, %f, %f}\n", name, vector.x(), vector.y(), vector.z(), vector.w());
}

__device__ bool intersect(const Rendering::Ray &ray, CudaArray<Cuda::Types::Triangle>::const_pointer *data, float &t, float &u, float &v)
{
	bool returnValue;
	float determinant, inverseDeterminant;
	
	Cuda::Types::Triangle triangle	= **data;
	Math::Vector4 v0o, pVector, qVector;
	
	// Sub
	v0o = ray.origin - triangle.v0;
	
	// Cross
	pVector = ray.direction.crossProduct(triangle.e02);
	qVector = v0o.crossProduct(triangle.e01);
	
	determinant = triangle.e01.dotProduct(pVector);
	inverseDeterminant = 1.0f / determinant;
	
	u = v0o.dotProduct(pVector) * inverseDeterminant;
	v = ray.direction.dotProduct(qVector) * inverseDeterminant;
	t = triangle.e02.dotProduct(qVector) * inverseDeterminant;
	(*data)++;
	
	// Conditions
	bool c0, c1, c2, c3, c4, c5;
	c0 = determinant < Math::epsilon;
	c1 = u < 0.0f;
	c2 = u > 1.0f;
	c3 = v < 0.0f;
	c4 = (u + v) > 1.0f;
	c5 = t > Math::epsilon;
	returnValue = !(c0 | c1 | c2 | c3 | c4) & c5;
	
	return returnValue;
}

__device__ float traceRay(const Rendering::Ray &ray, const Cuda::Types::Scene &scene, Cuda::Types::IntersectionInfo &intersection)
{
	float returnValue							= 0.0f;
	
	const Cuda::Types::Triangle *dataPointer	= scene.triangleBuffer;
	const Cuda::Types::Mesh *nearestMesh		= nullptr;
	uint32_t nearestTriangle					= 0xFFFFFFFF;
	float distance								= 1E7f;
	float newDistance							= distance;
//	float u = 0;
//	float v = 0;
	
	// Intersect triangles
	for (uint32_t triangleIndex = 0; triangleIndex < scene.triangleCount; triangleIndex++)
	{
		float u, v;
		bool intersected = intersect(ray, &dataPointer, newDistance, u, v);
		
		if ((newDistance < distance) & intersected)
		{
			nearestMesh		= &scene.meshBuffer[scene.triangleBuffer[triangleIndex].meshIndex];
			distance		= newDistance;
			nearestTriangle	= triangleIndex;
		}
	}
	
	returnValue					= distance;
	intersection.mesh			= nearestMesh;
	intersection.triangleOffset	= nearestTriangle;
//	intersection.u = u;
//	intersection.v = v;
	
	return returnValue;
}

__device__ void createCoordinateSystem(const Math::Vector4 &normal, Math::Vector4 &tangentNormal, Math::Vector4 &binormal)
{
	const Math::Vector4 a	= Math::Vector4{normal.z(), 0.0f, -normal.x()};
	const Math::Vector4 b	= Math::Vector4{0.0f, -normal.z(), normal.y()};
	float t					= fabsf(normal.x()) > fabsf(normal.y());
	
	tangentNormal			= Math::lerp(a, b, t).normalize();
	
	binormal				= normal.crossProduct(tangentNormal);
}

__device__ Math::Vector4 sampleUniform(const float r1, const float r2)
{
	float sinTheta	= sqrtf(1.0f - r1 * r1);
	float phi		= 2.0f * float(M_PI) * r2;
	float x			= sinTheta * cosf(phi);
	float z			= sinTheta * sinf(phi);
	
	return {x, r1, z};
}

__device__ Math::Vector4 sampleImportant(const float r1, const float r2, const float a)
{
	float theta		= acosf(sqrtf((1.0f - r1) / ((a * a - 1.0f) * r1 + 1.0f)));
	float phi		= 2.0f * float(M_PI) * r2;
	float x			= cosf(phi) * sinf(theta);
	float y			= cosf(theta);
	float z			= sinf(phi) * sinf(theta);
	
	return {x, y, z};
}

__device__ Math::Vector4 randomDirection(const Math::Vector4 &normal, hiprandState &rng, float &cosinusTheta, const float roughness)
{
//	float ratio;
	
	Math::Vector4 Nt;
	Math::Vector4 Nb;
	
	createCoordinateSystem(normal, Nt, Nb);
	
	// Generate hemisphere
//	cosinusTheta			= Math::pow(hiprand_uniform(&rng), 0.5f);
//	ratio					= hiprand_uniform(&rng);
	
//	Math::Vector4 sample	= sampleUniform(cosinusTheta, ratio);
	Math::Vector4 sample	= sampleImportant(hiprand_uniform(&rng), hiprand_uniform(&rng), roughness);
	
	Math::Matrix4x4 localToWorldMatrix{
		{Nb.x(), normal.x(), Nt.x()},
		{Nb.y(), normal.y(), Nt.y()},
		{Nb.z(), normal.z(), Nt.z()}
	};
	
	return (localToWorldMatrix * sample).normalize();
}

__device__ Math::Vector4 interpolateNormal(const Math::Vector4 &intersectionPoint, const Types::Triangle *data)
{
	Math::Vector4 returnValue, p, n0, n1, n2, n01, n02, v0, v1, v2, e01, e02, v12, v0p, v1p, v2p, vab, v2ab;
	
	v0	= data->v0;
	e01	= data->e01;
	e02	= data->e02;
	v1	= e01 + v0;
	v2	= e02 + v0;
	n0	= data->n0;
	n1	= data->n1;
	n2	= data->n2;
	
	data++;
	
	p	= intersectionPoint;
	v12	= v2 - v1;
	v0p	= p - v0;
	v1p	= p - v1;
	v2p	= p - v2;
	
	float a, denominator;
	
	denominator = (e01.x() * v2p.y() - v2p.x() * e01.y()) + Math::epsilon;
	a			= (-(v0.x() * v2p.y() - v2p.x() * v0.y() + v2p.x() * v2.y() - v2.x() * v2p.y())) / denominator;
//	b			= (e01.x() * v0.y() - e01.x() * v2.y() - v0.x() * e01.y() + v2.x() * e01.y()) / denominator;
	
	vab		= v0 + a * e01;
	
	n01		= Math::lerp(n1, n0, a).normalize();
	v2ab	= vab - v2;
	
	returnValue = Math::lerp(n01, n2, (v2p.magnitude() / v2ab.magnitude())).normalize();
	
	return returnValue;
}

__device__ Ray createCameraRay(const uint pixelX, const uint pixelY, const uint width, const uint height, const float fieldOfView, hiprandState &rng)
{
	float fovRadians	= fieldOfView / 180.0f * float(M_PI);
	float zCoordinate	= -(width/(2.0f * tanf(fovRadians / 2.0f)));
	
	float offsetX, offsetY;
	offsetX = hiprand_uniform(&rng)  - 0.5f;
	offsetY = hiprand_uniform(&rng) - 0.5f;
	
	float x = (pixelX + offsetX + 0.5f) - (width / 2.0f);
	float y = -(pixelY + offsetY + 0.5f) + (height / 2.0f);
	
	Math::Vector4 direction{x, y, zCoordinate};
	direction.normalize();
	
	return Ray{Math::Vector4{}, direction};
}

__global__ void castRay(const Cuda::Types::Tile tile, hiprandState *rngs, const uint32_t width, const uint32_t height, const float fieldOfView,
						const Cuda::Types::Scene scene, const size_t maxBounces, const Math::Vector4 skyColor, Math::Vector4 *pixels)
{
	const uint pixelX				= blockIdx.x * blockDim.x + threadIdx.x;
	const uint pixelY				= blockIdx.y * blockDim.y + threadIdx.y;
	const uint pixelIndex			= pixelY * width + pixelX;
	
	// Return early if pixel is outside tile
	if ((pixelX >= tile.x1) | (pixelY >= tile.y1))
	{
		return;
	}
	
	hiprandState rng					= rngs[pixelIndex];
	Ray ray							= createCameraRay(pixelX, pixelY, width, height, fieldOfView, rng);
	
	Math::Vector4 returnValue		= pixels[pixelIndex];
	Math::Vector4 mask				= {1.0f, 1.0f, 1.0f};
	
	Math::Vector4 currentDirection	= ray.direction;
	Math::Vector4 currentOrigin		= ray.origin;
	
	float cosinusTheta;
	
	for (size_t currentBounce = 0; currentBounce < maxBounces; currentBounce++)
	{
		Math::Vector4 intersectionPoint;
		Types::IntersectionInfo objectIntersection;
		Math::Vector4 normal;
		float objectDistance	= traceRay({currentOrigin, currentDirection}, scene, objectIntersection);
		
		intersectionPoint		= currentOrigin + (objectDistance * currentDirection);
		
		if (objectIntersection.mesh != nullptr)
		{
			const Cuda::Types::Triangle *dataPointer;
			Math::Vector4 objectColor, newDirection, diffuse, specular, kd, ks;
			Material objectMaterial		= scene.materialBuffer[objectIntersection.mesh->materialOffset];
			
			objectColor					= objectMaterial.color;
			
			// Calculate normal
			dataPointer					= scene.triangleBuffer + objectIntersection.triangleOffset;
			normal						= interpolateNormal(intersectionPoint, dataPointer);
			
			// Calculate new origin and offset
			currentOrigin				= intersectionPoint + (Math::epsilon * normal);
			
			// Global illumination
			newDirection				= randomDirection(normal, rng, cosinusTheta, objectMaterial.roughness * objectMaterial.roughness);
			cosinusTheta				= normal.dotProduct(newDirection);
			
			diffuse						= objectColor * Shader::diffuseLambert();
			specular					= Shader::specularCookTorrance(objectMaterial, -currentDirection, newDirection, normal, ks);
			kd							= (Math::Vector4{1.0f} - ks) * (1.0f - objectMaterial.metallic);
			
			currentDirection			= newDirection;
			
			returnValue					+= objectMaterial.emittance * objectColor * mask;
			mask						*= ((kd * diffuse) + specular) * cosinusTheta;
		}
		else
		{
			returnValue					+= skyColor * mask;
			break;
		}
	}
	
	rngs[pixelIndex]	= rng;
	pixels[pixelIndex]	= returnValue;
}

__global__ void setupRngs(hiprandState *rngs, const uint32_t seed, const Cuda::Types::Tile tile, const uint32_t width)
{
	const uint pixelX		= blockIdx.x * blockDim.x + threadIdx.x;
	const uint pixelY		= blockIdx.y * blockDim.y + threadIdx.y;
	const uint pixelIndex	= pixelY * width + pixelX;
	
	// Return early if pixel is outside tile
	if ((pixelX >= tile.x1) | (pixelY >= tile.y1))
	{
		return;
	}
	
	hiprand_init(seed, pixelIndex, 0, &rngs[pixelIndex]);
}

__global__ void initializePixels(Math::Vector4 *pixels, const Cuda::Types::Tile tile, const uint32_t width)
{
	const uint pixelX		= blockIdx.x * blockDim.x + threadIdx.x;
	const uint pixelY		= blockIdx.y * blockDim.y + threadIdx.y;
	const uint pixelIndex	= pixelY * width + pixelX;
	
	// Return early if pixel is outside tile
	if ((pixelX >= tile.x1) | (pixelY >= tile.y1))
	{
		return;
	}
	
	pixels[pixelIndex] = {};
}

__global__ void finalizePixels(Math::Vector4 *pixels, const Cuda::Types::Tile tile, const uint32_t width, const uint32_t samples)
{
	const uint pixelX		= blockIdx.x * blockDim.x + threadIdx.x;
	const uint pixelY		= blockIdx.y * blockDim.y + threadIdx.y;
	const uint pixelIndex	= pixelY * width + pixelX;
	
	// Return early if pixel is outside tile
	if ((pixelX >= tile.x1) | (pixelY >= tile.y1))
	{
		return;
	}
	
	pixels[pixelIndex] /= float(samples);
}

__global__ void encodeGamma(Math::Vector4 *pixels, const Cuda::Types::Tile tile, const uint32_t width)
{
	const uint pixelX		= blockIdx.x * blockDim.x + threadIdx.x;
	const uint pixelY		= blockIdx.y * blockDim.y + threadIdx.y;
	const uint pixelIndex	= pixelY * width + pixelX;
	
	// Return early if pixel is outside tile
	if ((pixelX >= tile.x1) | (pixelY >= tile.y1))
	{
		return;
	}
	
	pixels[pixelIndex] = Shader::encodeGamma(pixels[pixelIndex], 2.2f);
}

__host__ void cudaRender(
	FrameBuffer							&frameBuffer,
	RandomNumberGenerator				rng,
	const CudaArray<Types::Triangle>	&triangleBuffer,
	const CudaArray<Types::Mesh>		&meshBuffer,
	const CudaArray<Material>			&materialBuffer,
//	const AbstractRenderer::CallBack	&callback,
	const bool							&abort,
	const uint32_t						samples,
	const uint32_t						maxBounces,
	const uint32_t						tileSize,
	const float							fieldOfView,
	const Math::Vector4					&skyColor)
{
	const uint32_t pixelCount		= frameBuffer.width() * frameBuffer.height();
	const uint32_t threadsPerBlock	= 16;
	const uint32_t gridSizeX		= (frameBuffer.width() / threadsPerBlock) + 1;
	const uint32_t gridkSizeY		= (frameBuffer.height() / threadsPerBlock) + 1;
	
	Types::Scene scene{
		triangleBuffer.data(),
		triangleBuffer.size(),
		meshBuffer.data(),
		meshBuffer.size(),
		materialBuffer.data(),
		materialBuffer.size()
	};
	
	Types::Tile tile{0, 0, frameBuffer.width(), frameBuffer.height()};
	
	dim3 blockSize(threadsPerBlock, threadsPerBlock, 1);
	dim3 gridSize(gridSizeX, gridkSizeY, 1);
	
	CudaArray<hiprandState> rngBuffer(pixelCount);
	CudaArray<Math::Vector4> gpuFrameBuffer(pixelCount);
	
	setupRngs<<<gridSize, blockSize>>>(
		rngBuffer.data(),
		rng.get(),
		Types::Tile{0, 0, frameBuffer.width(), frameBuffer.height()},
		frameBuffer.width());
	hipDeviceSynchronize();
	handleCudaError(hipGetLastError());
	
	initializePixels<<<gridSize, blockSize>>>(
		gpuFrameBuffer.data(),
		tile,
		frameBuffer.width());
	hipDeviceSynchronize();
	handleCudaError(hipGetLastError());
	
//	const uint32_t tilesVertical	= frameBuffer.height() / tileSize + ((frameBuffer.height() % tileSize) > 0);
//	const uint32_t tilesHorizontal	= frameBuffer.width() / tileSize + ((frameBuffer.width() % tileSize) > 0);
	
//	for (uint32_t tileVertical = 0; tileVertical < tilesVertical; tileVertical++)
//	{
//		for (uint32_t tileHorizontal = 0; tileHorizontal < tilesHorizontal; tileHorizontal++)
//		{
//			uint32_t startVertical		= tileSize * tileVertical;
//			uint32_t startHorizontal	= tileSize * tileHorizontal;
//			uint32_t endVertical		= std::min(startVertical + tileSize, frameBuffer.height());
//			uint32_t endHorizontal		= std::min(startHorizontal + tileSize, frameBuffer.width());
			
//			tile = {startHorizontal, startVertical, endHorizontal, endVertical};
			
			for (uint32_t sample = 0; sample < samples; sample++)
			{
				castRay<<<gridSize, blockSize>>>(
					tile,
					rngBuffer.data(),
					frameBuffer.width(),
					frameBuffer.height(),
					fieldOfView,
					scene,
					maxBounces,
					skyColor,
					gpuFrameBuffer.data());
				hipDeviceSynchronize();
				handleCudaError(hipGetLastError());
			}
			
//			if (!abort)
//			{
//				callBack(startHorizontal, startVertical, endHorizontal, endVertical);
//			}
//		}
//	}
	
	finalizePixels<<<gridSize, blockSize>>>(
		gpuFrameBuffer.data(),
		Types::Tile{0, 0, frameBuffer.width(), frameBuffer.height()},
		frameBuffer.width(),
		samples);
	hipDeviceSynchronize();
	handleCudaError(hipGetLastError());
	
	encodeGamma<<<gridSize, blockSize>>>(
		gpuFrameBuffer.data(),
		Types::Tile{0, 0, frameBuffer.width(), frameBuffer.height()},
		frameBuffer.width());
	hipDeviceSynchronize();
	handleCudaError(hipGetLastError());
	
	// Copy frame buffer back
	frameBuffer = FrameBuffer::fromRawData(gpuFrameBuffer.data(), frameBuffer.width(), frameBuffer.height());
}

}
}
}
