#include "hip/hip_runtime.h"
#include "cuda/cudaarray.h"
#include "cuda/cudatypes.h"
#include "rendering/framebuffer.h"
#include "math/algorithms.h"
#include "math/matrix4x4.h"
#include "math/vector4.h"
#include "rendering/ray.h"
#include "rendering/randomnumbergenerator.h"

namespace ToyPT
{
namespace Rendering
{
namespace Cuda
{

__host__ void handleCudaError(const hipError_t error)
{
	if (error != hipSuccess)
	{
		printf("CUDA Error: %s %s\n", hipGetErrorName(error), hipGetErrorString(error));
	}
}

__device__ void print(const char *name, const Math::Vector4 &vector)
{
	printf("%s := {%f, %f, %f, %f}\n", name, vector.x(), vector.y(), vector.z(), vector.w());
}

__device__ bool intersect(const Rendering::Ray &ray, CudaArray<Cuda::Types::Triangle>::const_pointer *data, float &t, float &u, float &v)
{
	bool returnValue;
	float determinant, inverseDeterminant;
	
	Cuda::Types::Triangle triangle	= **data;
	Math::Vector4 v0o, pVector, qVector;
	
	// Sub
	v0o = ray.origin - triangle.v0;
	
	// Cross
	pVector = ray.direction.crossProduct(triangle.e02);
	qVector = v0o.crossProduct(triangle.e01);
	
	determinant = triangle.e01.dotProduct(pVector);
	inverseDeterminant = 1.0f / determinant;
	
	u = v0o.dotProduct(pVector) * inverseDeterminant;
	v = ray.direction.dotProduct(qVector) * inverseDeterminant;
	t = triangle.e02.dotProduct(qVector) * inverseDeterminant;
	(*data)++;
	
	// Conditions
	bool c0, c1, c2, c3, c4, c5;
	c0 = determinant < Math::epsilon;
	c1 = u < 0.0f;
	c2 = u > 1.0f;
	c3 = v < 0.0f;
	c4 = (u + v) > 1.0f;
	c5 = t > Math::epsilon;
	returnValue = !(c0 | c1 | c2 | c3 | c4) & c5;
	
	return returnValue;
}

__device__ float traceRay(const Rendering::Ray &ray, const Cuda::Types::Scene &scene, Cuda::Types::IntersectionInfo &intersection)
{
	float returnValue							= 0.0f;
	
	const Cuda::Types::Triangle *dataPointer	= scene.triangleBuffer;
	const Cuda::Types::Mesh *nearestMesh		= nullptr;
	uint32_t nearestTriangle					= 0xFFFFFFFF;
	float distance								= 1E7f;
	float newDistance							= distance;
//	float u = 0;
//	float v = 0;
	
	// Intersect triangles
	for (uint32_t triangleIndex = 0; triangleIndex < scene.triangleCount; triangleIndex++)
	{
//		auto &triangle = scene.triangleBuffer[triangleIndex];
		
//		printf("v0  := {%f, %f, %f}\n", triangle.v0.x(), triangle.v0.y(), triangle.v0.z());
//		printf("e01 := {%f, %f, %f}\n", triangle.e01.x(), triangle.e01.y(), triangle.e01.z());
//		printf("e02 := {%f, %f, %f}\n", triangle.e02.x(), triangle.e02.y(), triangle.e02.z());
//		printf("e12 := {%f, %f, %f}\n", triangle.e12.x(), triangle.e12.y(), triangle.e12.z());
		
		float t, u, v;
		bool intersected = intersect(ray, &dataPointer, t, u, v);
		
		if (intersected == true)
		{
			printf("intersection %f\n", t);
		}
		else
		{
			printf("no intersection %f\n", t);
		}
		
		if ((newDistance < distance) & (intersected == true))
		{
			nearestMesh		= &scene.meshBuffer[scene.triangleBuffer[triangleIndex].meshIndex];
			distance		= newDistance;
			nearestTriangle	= triangleIndex;
		}
	}
	
	returnValue					= distance;
	intersection.mesh			= nearestMesh;
	intersection.triangleOffset	= nearestTriangle;
//	intersection.u = u;
//	intersection.v = v;
	
	return returnValue;
}

__device__ void createCoordinateSystem(const Math::Vector4 &normal, Math::Vector4 &tangentNormal, Math::Vector4 &binormal)
{
	const Math::Vector4 a	= Math::Vector4{normal.z(), 0.0f, -normal.x()};
	const Math::Vector4 b	= Math::Vector4{0.0f, -normal.z(), normal.y()};
	float t					= fabsf(normal.x()) > fabsf(normal.y());
	
	tangentNormal			= Math::lerp(a, b, t).normalize();
	
	binormal				= normal.crossProduct(tangentNormal);
}

__device__ Math::Vector4 createUniformHemisphere(const float r1, const float r2)
{
	float sinTheta	= sqrtf(1.0f - r1 * r1);
	float phi		= 2.0f * float(M_PI) * r2;
	float x			= sinTheta * cosf(phi);
	float z			= sinTheta * sinf(phi);
	
	return {x, r1, z};
}

__device__ Math::Vector4 randomDirection(const Math::Vector4 &normal, RandomNumberGenerator &rng, float &cosinusTheta)
{
	float ratio;
	
	Math::Vector4 Nt;
	Math::Vector4 Nb;
	
	createCoordinateSystem(normal, Nt, Nb);
	
	// Generate hemisphere
	constexpr float scalingFactor	= 1.0f / float(0xFFFFFFFF);
	cosinusTheta					= std::pow(rng.get(scalingFactor), 0.5f);
	ratio							= rng.get(scalingFactor);
	
	Math::Vector4 sample			= createUniformHemisphere(cosinusTheta, ratio);
	
	Math::Matrix4x4 localToWorldMatrix{
		{Nb.x(), normal.x(), Nt.x()},
		{Nb.y(), normal.y(), Nt.y()},
		{Nb.z(), normal.z(), Nt.z()}
	};
	
	return (localToWorldMatrix * sample).normalize();
}

__device__ Math::Vector4 interpolateNormal(const Math::Vector4 &intersectionPoint, const Types::Triangle *data)
{
	Math::Vector4 returnValue, p, n0, n1, n2, n01, n02, v0, v1, v2, e01, e02, v12, v0p, v1p, v2p, vab, v2ab;
	
	v0	= data->v0;
	e01	= data->e01;
	e02	= data->e02;
	v1	= e01 + v0;
	v2	= e02 + v0;
	n0	= data->n0;
	n1	= data->n1;
	n2	= data->n2;
	
	data++;
	
	p	= intersectionPoint;
	v12	= v2 - v1;
	v0p	= p - v0;
	v1p	= p - v1;
	v2p	= p - v2;
	
	float a, denominator;
	
	denominator = (e01.x() * v2p.y() - v2p.x() * e01.y()) + Math::epsilon;
	a			= (-(v0.x() * v2p.y() - v2p.x() * v0.y() + v2p.x() * v2.y() - v2.x() * v2p.y())) / denominator;
//	b			= (e01.x() * v0.y() - e01.x() * v2.y() - v0.x() * e01.y() + v2.x() * e01.y()) / denominator;
	
	vab		= v0 + a * e01;
	
	n01		= Math::lerp(n1, n0, a).normalize();
	v2ab	= vab - v2;
	
	returnValue = Math::lerp(n01, n2, (v2p.magnitude() / v2ab.magnitude())).normalize();
	
	return returnValue;
}

__device__ Ray createCameraRay(const uint pixelX, const uint pixelY, const uint width, const uint height, const float fieldOfView, RandomNumberGenerator &rng)
{
	float fovRadians = fieldOfView / 180.0f * float(M_PI);
	float zCoordinate = -(width/(2.0f * tanf(fovRadians / 2.0f)));
	
	float offsetX, offsetY;
	constexpr float scalingFactor = 1.0f / float(0xFFFFFFFF);
	offsetX = rng.get(scalingFactor)  - 0.5f;
	offsetY = rng.get(scalingFactor) - 0.5f;
	
	float x = (pixelX + offsetX + 0.5f) - (width / 2.0f);
	float y = -(pixelY + offsetY + 0.5f) + (height / 2.0f);
	
	Math::Vector4 direction{x, y, zCoordinate};
	direction.normalize();
	
	return Ray{Math::Vector4{}, direction};
}

__global__ void castRay(const Cuda::Types::Tile tile, RandomNumberGenerator *rngs, const uint32_t width, const uint32_t height, const float fieldOfView,
						const Cuda::Types::Scene scene, const size_t maxBounces, const Math::Vector4 skyColor, Math::Vector4 *pixels)
{
	const uint pixelX		= blockIdx.x * blockDim.x + threadIdx.x;
	const uint pixelY		= blockIdx.y * blockDim.y + threadIdx.y;
	const uint pixelIndex	= pixelY * width + pixelX;
	
	// Return early if pixel is outside tile
	if ((pixelX >= tile.x1) | (pixelY >= tile.y1))
	{
		return;
	}
	
	RandomNumberGenerator &rng = rngs[pixelIndex];
	Ray ray = createCameraRay(pixelX, pixelY, width, height, fieldOfView, rng);
	
	Math::Vector4 returnValue = {0.0f, 0.0f, 0.0f};
	Math::Vector4 mask = {1.0f, 1.0f, 1.0f};
	
	Math::Vector4 currentDirection = ray.direction;
	Math::Vector4 currentOrigin = ray.origin;
	
	float cosinusTheta;
	
//	for (size_t currentBounce = 0; currentBounce < maxBounces; currentBounce++)
	{
		Math::Vector4 intersectionPoint;
		Types::IntersectionInfo objectIntersection;
		Math::Vector4 normal;
		float objectDistance = traceRay({currentOrigin, currentDirection}, scene, objectIntersection);
		
		intersectionPoint = currentOrigin + (objectDistance * currentDirection);
		
		if (objectIntersection.mesh != nullptr)
		{
			Material objectMaterial = scene.materialBuffer[objectIntersection.mesh->materialOffset];
			Math::Vector4 objectColor = objectMaterial.color;
			
			// Calculate normal
			const Cuda::Types::Triangle *dataPointer = scene.triangleBuffer + objectIntersection.triangleOffset;
			normal = interpolateNormal(intersectionPoint, dataPointer);
			
			// Calculate new origin and offset
//			currentOrigin = intersectionPoint + (Math::epsilon * normal);
			
//			// Global illumination
//			Math::Vector4 newDirection, reflectedDirection, diffuseDirection;
//			Math::Vector4 diffuse, specular;
			
//			diffuseDirection = randomDirection(normal, rng, cosinusTheta);
//			reflectedDirection = (currentDirection - 2.0f * currentDirection.dotProduct(normal) * normal).normalize();
			
//			newDirection = Math::lerp(diffuseDirection, reflectedDirection, objectMaterial.roughness);
			
//			specular = Math::Vector4{1.0f, 1.0f, 1.0f} * (1.0f - objectMaterial.roughness);
//			diffuse = Math::Vector4{1.0f, 1.0f, 1.0f} - specular;
			
//			currentDirection = newDirection;
			
//			returnValue += objectMaterial.emittance * mask;
//			mask *= (2.0f * objectColor * diffuse + specular) * cosinusTheta;
			returnValue = objectColor;
//			returnValue = {0.0f, 1.0f, 0.0f};
		}
		else
		{
//			returnValue = {1.0f, 0.0f, 0.0f};
			returnValue += skyColor * mask;
//			break;
		}
	}
	
	pixels[pixelIndex] = returnValue;
}

__host__ void cudaRender(
	FrameBuffer &frameBuffer, RandomNumberGenerator rng,
	const CudaArray<Types::Triangle> &triangleBuffer,
	const CudaArray<Types::Mesh> &meshBuffer,
	const CudaArray<Material> &materialBuffer,
	const uint32_t samples,
	const uint32_t maxBounces,
	const float fieldOfView,
	const Math::Vector4 &skyColor)
{
	const uint32_t pixelCount		= frameBuffer.width() * frameBuffer.height();
	const uint32_t threadsPerBlock	= 32;
	const uint32_t gridSizeX		= (frameBuffer.width() / threadsPerBlock) + 1;
	const uint32_t gridkSizeY		= (frameBuffer.height() / threadsPerBlock) + 1;
	
	Types::Scene scene{
		triangleBuffer.data(),
		triangleBuffer.size(),
		meshBuffer.data(),
		meshBuffer.size(),
		materialBuffer.data(),
		materialBuffer.size()
	};
	
	dim3 blockSize(threadsPerBlock, threadsPerBlock, 1);
	dim3 gridSize(gridSizeX, gridkSizeY, 1);
	
	printf("width=%u height=%u\n", frameBuffer.width(), frameBuffer.height());
	
	CudaArray<RandomNumberGenerator> rngs(pixelCount);
	
	for (uint32_t i = 0; i < rngs.size(); i++)
	{
		rngs[i] = rng.get();
	}
	
	printf("created rngs\n");
	
	CudaArray<Math::Vector4> gpuFrameBuffer(pixelCount);
	
	printf("allocated GPU framebuffer... launching kernel...\n");
	
	printf("gridSize=(%u, %u, %u) blockSize=(%u, %u, %u)\n", gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z);
	
	castRay<<<gridSize, blockSize>>>(
		Types::Tile{0, 0, frameBuffer.width(), frameBuffer.height()},
		rngs.data(),
		frameBuffer.width(),
		frameBuffer.height(),
		fieldOfView,
		scene,
		maxBounces,
		skyColor,
		gpuFrameBuffer.data());
	
	hipDeviceSynchronize();
	
	handleCudaError(hipGetLastError());
	
	printf("finished kernel execution\n");
	
	// Copy frame buffer back
	frameBuffer = FrameBuffer::fromRawData(gpuFrameBuffer.data(), frameBuffer.width(), frameBuffer.height());
}

}
}
}
