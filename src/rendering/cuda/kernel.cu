#include "hip/hip_runtime.h"
#include "cuda/cudaarray.h"
#include "cuda/cudatypes.h"
#include "rendering/framebuffer.h"
#include "math/algorithms.h"
#include "math/matrix4x4.h"
#include "math/vector4.h"
#include "rendering/ray.h"
#include "rendering/randomnumbergenerator.h"

namespace ToyPT
{
namespace Rendering
{
namespace Cuda
{

__device__ bool intersect(const Rendering::Ray &ray, CudaArray<Cuda::Types::Triangle>::const_pointer *data, float &t, float &u, float &v)
{
	bool returnValue;
	float determinant, inverseDeterminant;
	
	Math::Vector4 v0o, pVector, qVector;
	
	// Sub
	v0o = ray.origin - (*data)->v0;
	
	// Cross
	pVector = ray.direction.crossProduct((*data)->e02);
	qVector = v0o.crossProduct((*data)->e01);
	
	determinant = (*data)->e01.dotProduct(pVector);
	inverseDeterminant = 1.0f / determinant;
	
	u = v0o.dotProduct(pVector) * inverseDeterminant;
	v = ray.direction.dotProduct(qVector) * inverseDeterminant;
	t = (*data)->e02.dotProduct(qVector) * inverseDeterminant;
	(*data)++;
	
	// Conditions
	bool c0, c1, c2, c3, c4, c5;
	c0 = determinant < Math::epsilon;
	c1 = u < 0.0f;
	c2 = u > 1.0f;
	c3 = v < 0.0f;
	c4 = (u + v) > 1.0f;
	c5 = t > Math::epsilon;
	returnValue = (c0 | c1 | c2 | c3 | c4) & c5;
	
	return returnValue;
}

__device__ float traceRay(const Rendering::Ray &ray, const Cuda::Types::Scene &scene, Cuda::Types::IntersectionInfo &intersection)
{
	float returnValue = 0.0f;
	
	const Cuda::Types::Triangle *dataPointer = scene.triangleBuffer;
	const Cuda::Types::Mesh *nearestMesh = nullptr;
	uint32_t nearestTriangle = 0xFFFFFFFF;
	float newDistance = 0.0f;
	float distance = 1E7f;
//	float u = 0;
//	float v = 0;
	newDistance = distance;
	
	// Intersect triangles
	for (uint32_t triangleIndex = 0; triangleIndex < scene.triangleCount; triangleIndex++)
	{
		float t, u, v;
		bool intersected = intersect(ray, &dataPointer, t, u, v);
		
		if ((newDistance < distance) & bool(intersected))
		{
			nearestMesh = &scene.meshBuffer[scene.triangleBuffer[triangleIndex].meshIndex];
			distance = newDistance;
			nearestTriangle = triangleIndex;
		}
	}
	
	returnValue = distance;
	intersection.mesh = nearestMesh;
	intersection.triangleOffset = nearestTriangle;
//	intersection.u = u;
//	intersection.v = v;
	
	return returnValue;
}

__device__ void createCoordinateSystem(const Math::Vector4 &normal, Math::Vector4 &tangentNormal, Math::Vector4 &binormal)
{
	const Math::Vector4 a = Math::Vector4{normal.z(), 0.0f, -normal.x()};
	const Math::Vector4 b = Math::Vector4{0.0f, -normal.z(), normal.y()};
	float t = fabsf(normal.x()) > fabsf(normal.y());
	
	tangentNormal = Math::lerp(a, b, t).normalize();
	
	binormal = normal.crossProduct(tangentNormal);
}

__device__ Math::Vector4 createUniformHemisphere(const float r1, const float r2)
{
	float sinTheta = sqrtf(1.0f - r1 * r1);
	float phi = 2.0f * float(M_PI) * r2;
	float x = sinTheta * cosf(phi);
	float z = sinTheta * sinf(phi);
	return {x, r1, z};
}

__device__ Math::Vector4 randomDirection(const Math::Vector4 &normal, RandomNumberGenerator &rng, float &cosinusTheta)
{
	float ratio;
	
	Math::Vector4 Nt;
	Math::Vector4 Nb;
	
	createCoordinateSystem(normal, Nt, Nb);
	
	// Generate hemisphere
	constexpr float scalingFactor = 1.0f / float(0xFFFFFFFF);
	cosinusTheta = std::pow(rng.get(scalingFactor), 0.5f);
	ratio = rng.get(scalingFactor);
	
	Math::Vector4 sample = createUniformHemisphere(cosinusTheta, ratio);
	
	Math::Matrix4x4 localToWorldMatrix{
		{Nb.x(), normal.x(), Nt.x()},
		{Nb.y(), normal.y(), Nt.y()},
		{Nb.z(), normal.z(), Nt.z()}
	};
	
	return (localToWorldMatrix * sample).normalize();
}

__device__ Math::Vector4 interpolateNormal(const Math::Vector4 &intersectionPoint, const Types::Triangle *data)
{
	Math::Vector4 returnValue, p, n0, n1, n2, n01, n02, v0, v1, v2, e01, e02, v12, v0p, v1p, v2p, vab, v2ab;
	
	v0 = data->v0;
	e01 = data->e01;
	e02 = data->e02;
	v1 = e01 + v0;
	v2 = e02 + v0;
	n0 = data->n0;
	n1 = data->n1;
	n2 = data->n2;
	
	data++;
	
	p = intersectionPoint;
	v12 = v2 - v1;
	v0p = p - v0;
	v1p = p - v1;
	v2p = p - v2;
	
	float a, denominator;
	
	denominator = (e01.x() * v2p.y() - v2p.x() * e01.y()) + Math::epsilon;
	a = (-(v0.x() * v2p.y() - v2p.x() * v0.y() + v2p.x() * v2.y() - v2.x() * v2p.y())) / denominator;
//	b = (e01.x() * v0.y() - e01.x() * v2.y() - v0.x() * e01.y() + v2.x() * e01.y()) / denominator;
	
	vab = v0 + a * e01;
	
	n01 = Math::lerp(n1, n0, a).normalize();
	v2ab = vab - v2;
	
	returnValue = Math::lerp(n01, n2, (v2p.magnitude() / v2ab.magnitude())).normalize();
	
	return returnValue;
}

__device__ Ray createCameraRay(const uint pixelX, const uint pixelY, const uint width, const uint height, const float fieldOfView, RandomNumberGenerator &rng)
{
	float fovRadians = fieldOfView / 180.0f * float(M_PI);
	float zCoordinate = -(width/(2.0f * tanf(fovRadians / 2.0f)));
	
	float offsetX, offsetY;
	constexpr float scalingFactor = 1.0f / float(0xFFFFFFFF);
	offsetX = rng.get(scalingFactor)  - 0.5f;
	offsetY = rng.get(scalingFactor) - 0.5f;
	
	float x = (pixelX + offsetX + 0.5f) - (width / 2.0f);
	float y = -(pixelY + offsetY + 0.5f) + (height / 2.0f);
	
	Math::Vector4 direction{x, y, zCoordinate};
	direction.normalize();
	
	return Ray{Math::Vector4{}, direction};
}

__global__ void castRay(const Cuda::Types::Tile tile, RandomNumberGenerator *rngs, const uint32_t width, const uint32_t height, const float fieldOfView,
						const Cuda::Types::Scene scene, const size_t maxBounces, const Math::Vector4 skyColor, Math::Vector4 *pixels)
{
	uint pixelX = threadIdx.x * blockIdx.x * blockDim.x;
	uint pixelY = threadIdx.y * blockIdx.y * blockDim.y;
	
	// Return early if pixel is outside tile
	if ((pixelX > tile.x1) | (pixelY > tile.y1))
	{
		return;
	}
	
	uint pixelIndex = pixelX + pixelY * width;
	
	RandomNumberGenerator &rng = rngs[pixelIndex];
	Ray ray = createCameraRay(pixelX, pixelY, width, height, fieldOfView, rng);
	
	Math::Vector4 returnValue = {0.0f, 0.0f, 0.0f};
	Math::Vector4 mask = {1.0f, 1.0f, 1.0f};
	
	Math::Vector4 currentDirection = ray.direction;
	Math::Vector4 currentOrigin = ray.origin;
	
	float cosinusTheta;
	
	for (size_t currentBounce = 0; currentBounce < maxBounces; currentBounce++)
	{
		Math::Vector4 intersectionPoint;
		Types::IntersectionInfo objectIntersection;
		Math::Vector4 normal;
		float objectDistance = traceRay({currentOrigin, currentDirection}, scene, objectIntersection);
		
		intersectionPoint = currentOrigin + (objectDistance * currentDirection);
		
		if (objectIntersection.mesh != nullptr)
		{
			Material objectMaterial = scene.materialBuffer[objectIntersection.mesh->materialOffset];
			Math::Vector4 objectColor = objectMaterial.color;
			
			// Calculate normal
			const Cuda::Types::Triangle *dataPointer = scene.triangleBuffer + objectIntersection.triangleOffset;
			normal = interpolateNormal(intersectionPoint, dataPointer);
			
			// Calculate new origin and offset
			currentOrigin = intersectionPoint + (Math::epsilon * normal);
			
			// Global illumination
			Math::Vector4 newDirection, reflectedDirection, diffuseDirection;
			Math::Vector4 diffuse, specular;
			
			diffuseDirection = randomDirection(normal, rng, cosinusTheta);
			reflectedDirection = (currentDirection - 2.0f * currentDirection.dotProduct(normal) * normal).normalize();
			
			newDirection = Math::lerp(diffuseDirection, reflectedDirection, objectMaterial.roughness);
			
			specular = Math::Vector4{1.0f, 1.0f, 1.0f} * (1.0f - objectMaterial.roughness);
			diffuse = Math::Vector4{1.0f, 1.0f, 1.0f} - specular;
			
			currentDirection = newDirection;
			
			returnValue += objectMaterial.emittance * mask;
			mask *= (2.0f * objectColor * diffuse + specular) * cosinusTheta;
		}
		else
		{
			returnValue += skyColor * mask;
			break;
		}
	}
	
//	pixels[pixelIndex] = returnValue;
	Math::Vector4 &pixel = pixels[pixelIndex];
	atomicAdd(&pixel.data()[0], returnValue.data()[0]);
	atomicAdd(&pixel.data()[1], returnValue.data()[1]);
	atomicAdd(&pixel.data()[2], returnValue.data()[2]);
}

__host__ void render(FrameBuffer &frameBuffer, RandomNumberGenerator rng,
					 const CudaArray<Types::Triangle> &triangleBuffer,
					 const CudaArray<Types::Mesh> &meshBuffer,
					 const CudaArray<Material> &materialBuffer,
					 const uint32_t samples,
					 const uint32_t maxBounces,
					 const float fieldOfView,
					 const Math::Vector4 &skyColor)
{
	const uint32_t pixelCount = frameBuffer.width() * frameBuffer.height();
	
	Types::Scene scene{
		triangleBuffer.data(),
		triangleBuffer.size(),
		meshBuffer.data(),
		meshBuffer.size(),
		materialBuffer.data(),
		materialBuffer.size()
	};
	
	dim3 gridSize(samples);
	dim3 blockSize(frameBuffer.width(), frameBuffer.height());
	
	CudaArray<RandomNumberGenerator> rngs(pixelCount);
	
	for (uint32_t i = 0; i < rngs.size(); i++)
	{
		rngs[i] = rng.get();
	}
	
	CudaArray<Math::Vector4> gpuFrameBuffer(pixelCount);
	
	castRay<<<gridSize, blockSize>>>(Types::Tile{0, 0, frameBuffer.width(), frameBuffer.height()}, rngs.data(), frameBuffer.width(), frameBuffer.height(), fieldOfView, scene, maxBounces,
									 skyColor, gpuFrameBuffer.data());
	hipDeviceSynchronize();
	
	// Copy frame buffer back
	frameBuffer = FrameBuffer::fromRawData(gpuFrameBuffer.data(), frameBuffer.width(), frameBuffer.height());
}

}
}
}
