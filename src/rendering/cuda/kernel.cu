#include "hip/hip_runtime.h"
#include "cuda/cudaarray.h"
#include "cuda/cudatypes.h"
#include "math/algorithms.h"
#include "math/matrix4x4.h"
#include "math/vector4.h"
#include "ray.h"
#include "randomnumbergenerator.h"

namespace Rendering
{

namespace Cuda
{

__device__ bool intersect(const Rendering::Ray &ray, CudaArray<Cuda::Types::Triangle>::const_pointer *data, float &t, float &u, float &v)
{
	bool returnValue;
	float determinant, inverseDeterminant;
	
	Math::Vector4 v0o, pVector, qVector;
	
	// Sub
	v0o = ray.origin - (*data)->v0;
	
	// Cross
	pVector = ray.direction.crossProduct((*data)->e02);
	qVector = v0o.crossProduct((*data)->e01);
	
	determinant = (*data)->e01.dotProduct(pVector);
	inverseDeterminant = 1.0f / determinant;
	
	u = v0o.dotProduct(pVector) * inverseDeterminant;
	v = ray.direction.dotProduct(qVector) * inverseDeterminant;
	t = (*data)->e02.dotProduct(qVector) * inverseDeterminant;
	(*data)++;
	
	// Conditions
	bool c0, c1, c2, c3, c4, c5;
	c0 = determinant < Math::epsilon;
	c1 = u < 0.0f;
	c2 = u > 1.0f;
	c3 = v < 0.0f;
	c4 = (u + v) > 1.0f;
	c5 = t > Math::epsilon;
	returnValue = (c0 | c1 | c2 | c3 | c4) & c5;
	
	return returnValue;
}

__device__ float traceRay(const Rendering::Ray &ray, const Cuda::Types::Scene &scene, Cuda::Types::IntersectionInfo &intersection)
{
	float returnValue = 0.0f;
	
	const Cuda::Types::Triangle *dataPointer = scene.triangleBuffer;
	const Cuda::Types::Mesh *nearestMesh = nullptr;
	uint32_t nearestTriangle = 0xFFFFFFFF;
	float newDistance = 0.0f;
	float distance = 1E7f;
	float u = 0;
	float v = 0;
	newDistance = distance;
	
	// Intersect triangles
	for (uint32_t triangleIndex = 0; triangleIndex < scene.triangleCount; triangleIndex++)
	{
		float t, u, v;
		bool intersected = intersect(ray, &dataPointer, t, u, v);
		
		if ((newDistance < distance) & bool(intersected))
		{
			nearestMesh = &scene.meshBuffer[scene.triangleBuffer[triangleIndex].meshIndex];
			distance = newDistance;
			nearestTriangle = triangleIndex;
		}
	}
	
	returnValue = distance;
	intersection.mesh = nearestMesh;
	intersection.triangleOffset = nearestTriangle;
//	intersection.u = u;
//	intersection.v = v;
	
	return returnValue;
}

__device__ void createCoordinateSystem(const Math::Vector4 &normal, Math::Vector4 &tangentNormal, Math::Vector4 &binormal)
{
	const Math::Vector4 a = Math::Vector4{normal.z(), 0.0f, -normal.x()};
	const Math::Vector4 b = Math::Vector4{0.0f, -normal.z(), normal.y()};
	float t = fabsf(normal.x()) > fabsf(normal.y());
	
	tangentNormal = Math::lerp(a, b, t).normalize();
	
	binormal = normal.crossProduct(tangentNormal);
}

__device__ Math::Vector4 createUniformHemisphere(const float r1, const float r2)
{
	float sinTheta = sqrtf(1.0f - r1 * r1);
	float phi = 2.0f * float(M_PI) * r2;
	float x = sinTheta * cosf(phi);
	float z = sinTheta * sinf(phi);
	return {x, r1, z};
}

__device__ Math::Vector4 randomDirection(const Math::Vector4 &normal, RandomNumberGenerator &rng, float &cosinusTheta)
{
	float ratio;
	
	Math::Vector4 Nt;
	Math::Vector4 Nb;
	
	createCoordinateSystem(normal, Nt, Nb);
	
	// Generate hemisphere
	constexpr float scalingFactor = 1.0f / float(0xFFFFFFFF);
	cosinusTheta = std::pow(rng.get(scalingFactor), 0.5f);
	ratio = rng.get(scalingFactor);
	
	Math::Vector4 sample = createUniformHemisphere(cosinusTheta, ratio);
	
	Math::Matrix4x4 localToWorldMatrix{
		{Nb.x(), normal.x(), Nt.x()},
		{Nb.y(), normal.y(), Nt.y()},
		{Nb.z(), normal.z(), Nt.z()}
	};
	
	return (localToWorldMatrix * sample).normalize();
}

__device__ Math::Vector4 interpolateNormal(const Math::Vector4 &intersectionPoint, const Types::Triangle *data)
{
	Math::Vector4 returnValue, p, n0, n1, n2, n01, n02, v0, v1, v2, e01, e02, v12, v0p, v1p, v2p, vab, v2ab;
	
	v0 = data->v0;
	e01 = data->e01;
	e02 = data->e02;
	v1 = e01 + v0;
	v2 = e02 + v0;
	n0 = data->n0;
	n1 = data->n1;
	n2 = data->n2;
	
	data++;
	
	p = intersectionPoint;
	v12 = v2 - v1;
	v0p = p - v0;
	v1p = p - v1;
	v2p = p - v2;
	
	float a, b, denominator;
	
	denominator = (e01.x() * v2p.y() - v2p.x() * e01.y()) + Math::epsilon;
	a = (-(v0.x() * v2p.y() - v2p.x() * v0.y() + v2p.x() * v2.y() - v2.x() * v2p.y())) / denominator;
	b = (e01.x() * v0.y() - e01.x() * v2.y() - v0.x() * e01.y() + v2.x() * e01.y()) / denominator;
	
	vab = v0 + a * e01;
	
	n01 = Math::lerp(n1, n0, a).normalize();
	v2ab = vab - v2;
	
	returnValue = Math::lerp(n01, n2, (v2p.magnitude() / v2ab.magnitude())).normalize();
	
	return returnValue;
}

__global__ void castRay(const Rendering::Ray &ray, const Cuda::Types::Scene &scene, const size_t maxBounces,
						const Math::Vector4 &skyColor)
{
	Rendering::RandomNumberGenerator rng;
	Math::Vector4 returnValue = {0.0f, 0.0f, 0.0f};
	Math::Vector4 mask = {1.0f, 1.0f, 1.0f};
	
	Math::Vector4 currentDirection = ray.direction;
	Math::Vector4 currentOrigin = ray.origin;
	
	float cosinusTheta;
	
	for (size_t currentBounce = 0; currentBounce < maxBounces; currentBounce++)
	{
		Math::Vector4 intersectionPoint;
		Types::IntersectionInfo objectIntersection;
		Math::Vector4 normal;
		float objectDistance = traceRay({currentOrigin, currentDirection}, scene, objectIntersection);
		
		intersectionPoint = currentOrigin + (objectDistance * currentDirection);
		
		if (objectIntersection.mesh != nullptr)
		{
			Material objectMaterial = scene.materialBuffer[objectIntersection.mesh->materialOffset];
			Math::Vector4 objectColor = objectMaterial.color;
			
			// Calculate normal
			const Cuda::Types::Triangle *dataPointer = scene.triangleBuffer + objectIntersection.triangleOffset;
			normal = interpolateNormal(intersectionPoint, dataPointer);
			
			// Calculate new origin and offset
			currentOrigin = intersectionPoint + (Math::epsilon * normal);
			
			// Global illumination
			Math::Vector4 newDirection, reflectedDirection, diffuseDirection;
			Math::Vector4 diffuse, specular;
			
			diffuseDirection = randomDirection(normal, rng, cosinusTheta);
			reflectedDirection = (currentDirection - 2.0f * currentDirection.dotProduct(normal) * normal).normalize();
			
			newDirection = Math::lerp(diffuseDirection, reflectedDirection, objectMaterial.roughness);
			
			specular = Math::Vector4{1.0f, 1.0f, 1.0f} * (1.0f - objectMaterial.roughness);
			diffuse = Math::Vector4{1.0f, 1.0f, 1.0f} - specular;
			
			currentDirection = newDirection;
			
			returnValue += objectMaterial.emittance * mask;
			mask *= (2.0f * objectColor * diffuse + specular) * cosinusTheta;
		}
		else
		{
			returnValue += skyColor * mask;
			break;
		}
	}
	
//	return returnValue;
}

}
}
